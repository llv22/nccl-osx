// P2P Test by Greg Gutmann
 

#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdint.h"
 
int main()
{
    // GPUs
    int gpuid_0 = 0;
    int gpuid_1 = 1;
 
    // Memory Copy Size
    uint32_t size = pow(2, 26); // 2^26 = 67MB
 
    // Allocate Memory
    uint32_t* dev_0;
    hipSetDevice(gpuid_0);
    hipMalloc((void**)&dev_0, size);
 
    uint32_t* dev_1;
    hipSetDevice(gpuid_1);
    hipMalloc((void**)&dev_1, size);
 
    //Check for peer access between participating GPUs: 
    int can_access_peer_0_1;
    int can_access_peer_1_0;
    hipDeviceCanAccessPeer(&can_access_peer_0_1, gpuid_0, gpuid_1);
    hipDeviceCanAccessPeer(&can_access_peer_1_0, gpuid_1, gpuid_0);
    printf("cudaDeviceCanAccessPeer(%d->%d): %d\n", gpuid_0, gpuid_1, can_access_peer_0_1);
    printf("cudaDeviceCanAccessPeer(%d->%d): %d\n", gpuid_1, gpuid_0, can_access_peer_1_0);
 
    if (can_access_peer_0_1 && can_access_peer_1_0) {
        // Enable P2P Access
        hipSetDevice(gpuid_0);
        hipDeviceEnablePeerAccess(gpuid_1, 0);
        hipSetDevice(gpuid_1);
        hipDeviceEnablePeerAccess(gpuid_0, 0);
    }
 
    // Init Timing Data
    uint32_t repeat = 10;
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
 
    // Init Stream
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
 
    // ~~ Start Test ~~
    hipEventRecord(start, stream);
 
    //Do a P2P memcpy
    for (int i = 0; i < repeat; ++i) {
        hipMemcpyAsync(dev_0, dev_1, size, hipMemcpyDeviceToDevice, stream);
    }
 
    hipEventRecord(stop, stream);
    hipStreamSynchronize(stream);
    // ~~ End of Test ~~
 
    // Check Timing & Performance
    float time_ms;
    hipEventElapsedTime(&time_ms, start, stop);
    double time_s = time_ms / 1e3;
 
    double gb = size * repeat / (double)1e9;
    double bandwidth = gb / time_s;
 
    printf("Seconds: %f\n", time_s);
    printf("Unidirectional Bandwidth: %f (GB/s)\n", bandwidth);
 
    if (can_access_peer_0_1 && can_access_peer_1_0) {
        // Shutdown P2P Settings
        hipSetDevice(gpuid_0);
        hipDeviceDisablePeerAccess(gpuid_1);
        hipSetDevice(gpuid_1);
        hipDeviceDisablePeerAccess(gpuid_0);
    }
 
    // Clean Up
    hipFree(dev_0);
    hipFree(dev_1);
 
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);
}
